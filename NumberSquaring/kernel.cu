
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t squareWithCuda(long *input, long *output, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void square(long *d_out, long *d_in) {
	int idx = threadIdx.x;
	long f = d_in[idx];
	d_out[idx] = f * f;
}

int main()
{

	const int ARRAY_SIZE = 1024;

	long h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = long(i);
	}

	long h_out[ARRAY_SIZE] = { 0 };

	hipError_t cudaStatus = squareWithCuda(h_in, h_out, ARRAY_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Square With Cuda failed!");
		cin.get();
		return 1;
	}

	for (int i = 0; i < ARRAY_SIZE; i++) {
		printf("%d", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	/*
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
		*/

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	cin.get();

    return 0;
}

hipError_t squareWithCuda(long *input, long *output, unsigned int size) {
	long *dev_input = 0;
	long *dev_output = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_input, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, size * sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_output, output, size * sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	square<<<1.0, size>>>(dev_output, dev_input);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, size * sizeof(long), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_input);
	hipFree(dev_output);

	return cudaStatus;

}